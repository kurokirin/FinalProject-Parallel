#include "hip/hip_runtime.h"

#include <iostream>
#include <string>
#include <cstdlib>
#include <cstdint>
#include <map>
#include <fstream>
#include <vector>
#include <sstream>
#include <cmath>
#include <functional>
#include <set>
#include <ctime>

#define n 1024
#define T 256

using namespace std;
map<int, vector<int>> inlinks;
map<int, vector<int>> outlinks;
vector<int> sink;
map<int, double> pr_score;
int countConverge = 0;
double d = 0.85; //pagerank damping/teleportation factor in this project we use 0.85
double previous_perplex = 0;
int allPages = 0;


void pageArr(vector<std::string> str) {
	vector<int> pageIDs;
	vector<int> out;
	int count = 0, firstCol;
	for (string s : str) {
		stringstream page(s);
		int x = 0;
		page >> x;
		if (count == 0) {
			firstCol = x;
		}
		else {
			pageIDs.push_back(x);
		}
		count++;
	}
	inlinks.insert({ firstCol,pageIDs });
	outlinks.insert({ firstCol, out });
	allPages++;
}
std::vector<std::string> split(std::string strToSplit, char delimeter)
{
	std::stringstream ss(strToSplit);
	std::string item;
	std::vector<std::string> splittedStrings;
	while (std::getline(ss, item, delimeter))
	{
		splittedStrings.push_back(item);
	}
	return splittedStrings;
}
void readfile() {
	std::ifstream myfile;
	myfile.open("D:\\Parallel\\Parallel\\x64\\Debug\\citeseer.dat");
	string myText;
	cout << "Reading from the file" << endl;
	while (getline(myfile, myText)) {
		//cout << myText <<endl;
		pageArr(split(myText, ' '));
	}
	myfile.close();
	cout << "finish reading from file" << endl;
	//cin.get();
}

void findOutlinks() {
	cout << "find outlinks" << endl;
	for (auto& x : inlinks) {
		for (int i : x.second) {
			try {
				/*if (outlinks.at(i).empty()) {
					continue;
				}*/
				outlinks[i].push_back(x.first);
			}
			catch (const std::out_of_range& oor) {
				std::cerr << "Out of Range error: " << oor.what() << '\n';
				//cout << "pid: " << i << endl;
			}
		}
	}
}

void findSinkNode() {
	cout << "find sink node" << endl;
	for (auto& x : outlinks) {
		if (x.second.size() == 0) {
			sink.push_back(x.first);
		}
	}
}

void initialize() {
	cout << "initialize" << endl;
	double init_val = 1 / allPages;
	for (auto& x : inlinks) {
		pr_score.insert({ x.first, init_val });
	}

}
__global__ void calculateXxLog2X(double *input)
{
	int i = threadIdx.x;
	//c[i] = a[i] + b[i];
	input[i] = input[i] * log2(input[i]);
}
double getPerplexity() {
	cout << "get perplexity" << endl;
	double sum = 0;
	size_t pr_score_size = pr_score.size();
	double* hostBuffer = (double*)malloc(pr_score_size*sizeof(double));
	double* devBuffer;
	hipMalloc((void**)&devBuffer,pr_score_size);
	int i=0;
	for (auto& x : pr_score) {
		hostBuffer[i] = x.second;
		i++;
	}
	hipMemcpy(devBuffer, hostBuffer, pr_score_size,hipMemcpyHostToDevice);
	int nblocks = n / T;
	calculateXxLog2X<<<nblocks,T>>> (devBuffer);
	hipMemcpy(hostBuffer, devBuffer, pr_score_size, hipMemcpyDeviceToHost);
	for (i = 0; i < pr_score_size; i++) {
		sum += hostBuffer[i];
	}
	if (!isnormal(sum)) {
		cout << sum;
	}
	sum = pow(2, -(sum));
	return sum;
}

bool isConverge() {
	if (countConverge == 3) return true;
	else return false;
}

void runPageRank() {
	cout << "run pagerank" << endl;
	double sinkPR, temp = 0;
	double newPR;
	//vector<double> perplexities;
	for (int pid : sink) {
		try {
			temp += pr_score[pid];
		}
		catch (const std::out_of_range& oor) {
			std::cerr << "Out of Range error: " << oor.what() << '\n';
			//cout << "(sink)pid: " << pid << endl;
		}
	}
	while (!isConverge()) {
		//pr score of each page
		for (auto& pid : inlinks) {
			sinkPR = temp;
			newPR = (1 - d) / inlinks.size();
			newPR += (d*(sinkPR / inlinks.size()));
			//cout << "newPR1 = " << newPR << endl;
			for (int i : pid.second) {
				if (!outlinks[i].size() == 0) {
					newPR += (d*(pr_score[i] / outlinks[i].size()));
				}
				
			}
			//cout << "newPR2 = " << newPR << endl;
			pr_score[pid.first] = newPR;

		}
		double ceilPrePerPlex = ceil(getPerplexity());
		if (previous_perplex == ceilPrePerPlex) countConverge++;
		else {
			previous_perplex = ceilPrePerPlex;
			countConverge = 0;
		}
		//perplexities.push_back(getPerplexity());
		//cout << "perplexity = " << getPerplexity() << endl;
		//cout << "end round" << endl;
	}
}

vector<int> getRankedPages(int K) {
	cout << "get rank" << endl;
	map<double, int> rankmap;
	multimap<int, double> ::iterator iter;
	for (iter = pr_score.begin(); iter != pr_score.end(); iter++)
	{
		rankmap.insert({ (*iter).second, (*iter).first });
		//cout << iter->second << ": " << iter->first << endl;
	}

	vector<int> rank;
	int count = 0;
	for (auto& x : rankmap) {
		rank.push_back(x.second);
		count++;
		if (count == (K)) break;
	}
	return rank;
}

int main() {
	std::clock_t start, stop;
	start = std::clock();

	readfile();
	findOutlinks();
	findSinkNode();
	initialize();
	runPageRank();
	vector<int> rankpages = getRankedPages(100);

	stop = std::clock();
	double duration = stop - start / 1000.0;

	cout << "Top 100 pages are\n";
	for (int i = 0; i < 100; i++) {
		cout << "Rank#" << i << " PageID: " << rankpages[i] << "\n";
	}
	cout << "time: " << duration << endl;
	cin.get();
	return 0;
}